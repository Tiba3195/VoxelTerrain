#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <>
#include <>
#include <hip/hip_texture_types.h>
#include <builtin_types.h>
#include <vector_functions.h> 
#include <float.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>

#define POISSON_DISC_LEN 32
#define AMBIGOUS_LEN 60
#define THREADBLOCK_SIZE 256

typedef unsigned int uint;

__device__ float poissonDisc[POISSON_DISC_LEN][3] =
{
	{ 0.7768153f, 0.3749168f, -0.5059598f },
	{ 0.08306061f, 0.9473661f, -0.3091901 },
	{ 0.6623104f, 0.7395632f, 0.1199641 },
	{ 0.9948989f, 0.0497775f, 0.08774123 },
	{ 0.104239f, 0.2789151f, -0.9546416 },
	{ 0.5960904f, 0.01746058f, -0.8027275 },
	{ 0.4458466f, 0.1886109f, 0.8750125 },
	{ -0.07843895f, 0.4710891f, 0.8785911 },
	{ -0.3749092f, 0.9266203f, 0.02859987 },
	{ 0.1367656f, 0.9223449f, 0.3613518 },
	{ 0.8283083f, 0.01616495f, 0.5600392 },
	{ -0.607545f, 0.08607148f, 0.7896079 },
	{ -0.8451187f, 0.3715429f, 0.384357 },
	{ -0.9599981f, 0.1915317f, -0.2042528 },
	{ -0.3972329f, 0.09472971f, -0.9128156 },
	{ -0.6823229f, 0.4382687f, -0.585112 },
	{ 0.2192558f, -0.6357883f, -0.7400677 },
	{ 0.78632f, -0.4650563f, -0.406723 },
	{ 0.1986186f, -0.9779121f, 0.065104 },
	{ -0.5403743f, -0.5149913f, -0.6654168 },
	{ -0.4253772f, -0.9048665f, 0.0164598 },
	{ 0.642599f, -0.01937828f, -0.7659575 },
	{ -0.2056696f, -0.1125926f, -0.9721229 },
	{ 0.9760811f, -0.1599507f, 0.1472464 },
	{ 0.2192492f, -0.03539763f, -0.9750266 },
	{ 0.5748524f, -0.5088353f, 0.6408051 },
	{ -0.1517765f, -0.2752149f, 0.9493264 },
	{ -0.5343058f, -0.5594884f, 0.6336324 },
	{ -0.9227477f, -0.1243654f, 0.3647878 },
	{ -0.9137639f, -0.3176999f, -0.2531843 },
	{ 0.4282694f, -0.02752976f, 0.9032317 },
	{ 0.8152075f, -0.0009064535f, 0.5791682f }
};

__device__ int faces[] =
{
	-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 8, 3, 1, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 2, 11, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	2, 8, 3, 2, 11, 8, 11, 9, 8, -1, -1, -1, -1, -1, -1,
	3, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 10, 2, 8, 10, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 9, 0, 2, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 10, 2, 1, 9, 10, 9, 8, 10, -1, -1, -1, -1, -1, -1,
	3, 11, 1, 10, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 11, 1, 0, 8, 11, 8, 10, 11, -1, -1, -1, -1, -1, -1,
	3, 9, 0, 3, 10, 9, 10, 11, 9, -1, -1, -1, -1, -1, -1,
	9, 8, 11, 11, 8, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1,
	1, 2, 11, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 4, 7, 3, 0, 4, 1, 2, 11, -1, -1, -1, -1, -1, -1,
	9, 2, 11, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1,
	2, 11, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1,
	8, 4, 7, 3, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	10, 4, 7, 10, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1,
	9, 0, 1, 8, 4, 7, 2, 3, 10, -1, -1, -1, -1, -1, -1,
	4, 7, 10, 9, 4, 10, 9, 10, 2, 9, 2, 1, -1, -1, -1,
	3, 11, 1, 3, 10, 11, 7, 8, 4, -1, -1, -1, -1, -1, -1,
	1, 10, 11, 1, 4, 10, 1, 0, 4, 7, 10, 4, -1, -1, -1,
	4, 7, 8, 9, 0, 10, 9, 10, 11, 10, 0, 3, -1, -1, -1,
	4, 7, 10, 4, 10, 9, 9, 10, 11, -1, -1, -1, -1, -1, -1,
	9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1,
	1, 2, 11, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 0, 8, 1, 2, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1,
	5, 2, 11, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1,
	2, 11, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1,
	9, 5, 4, 2, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 10, 2, 0, 8, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1,
	0, 5, 4, 0, 1, 5, 2, 3, 10, -1, -1, -1, -1, -1, -1,
	2, 1, 5, 2, 5, 8, 2, 8, 10, 4, 8, 5, -1, -1, -1,
	11, 3, 10, 11, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1,
	4, 9, 5, 0, 8, 1, 8, 11, 1, 8, 10, 11, -1, -1, -1,
	5, 4, 0, 5, 0, 10, 5, 10, 11, 10, 0, 3, -1, -1, -1,
	5, 4, 8, 5, 8, 11, 11, 8, 10, -1, -1, -1, -1, -1, -1,
	9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1,
	0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1,
	1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 7, 8, 9, 5, 7, 11, 1, 2, -1, -1, -1, -1, -1, -1,
	11, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1,
	8, 0, 2, 8, 2, 5, 8, 5, 7, 11, 5, 2, -1, -1, -1,
	2, 11, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1,
	7, 9, 5, 7, 8, 9, 3, 10, 2, -1, -1, -1, -1, -1, -1,
	9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 10, -1, -1, -1,
	2, 3, 10, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1,
	10, 2, 1, 10, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1,
	9, 5, 8, 8, 5, 7, 11, 1, 3, 11, 3, 10, -1, -1, -1,
	5, 7, 0, 5, 0, 9, 7, 10, 0, 1, 0, 11, 10, 11, 0,
	10, 11, 0, 10, 0, 3, 11, 5, 0, 8, 0, 7, 5, 7, 0,
	10, 11, 5, 7, 10, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	11, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 8, 3, 5, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 0, 1, 5, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 8, 3, 1, 9, 8, 5, 11, 6, -1, -1, -1, -1, -1, -1,
	1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1,
	9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1,
	5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1,
	2, 3, 10, 11, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	10, 0, 8, 10, 2, 0, 11, 6, 5, -1, -1, -1, -1, -1, -1,
	0, 1, 9, 2, 3, 10, 5, 11, 6, -1, -1, -1, -1, -1, -1,
	5, 11, 6, 1, 9, 2, 9, 10, 2, 9, 8, 10, -1, -1, -1,
	6, 3, 10, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1,
	0, 8, 10, 0, 10, 5, 0, 5, 1, 5, 10, 6, -1, -1, -1,
	3, 10, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1,
	6, 5, 9, 6, 9, 10, 10, 9, 8, -1, -1, -1, -1, -1, -1,
	5, 11, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 3, 0, 4, 7, 3, 6, 5, 11, -1, -1, -1, -1, -1, -1,
	1, 9, 0, 5, 11, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1,
	11, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1,
	6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1,
	1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1,
	8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1,
	7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9,
	3, 10, 2, 7, 8, 4, 11, 6, 5, -1, -1, -1, -1, -1, -1,
	5, 11, 6, 4, 7, 2, 4, 2, 0, 2, 7, 10, -1, -1, -1,
	0, 1, 9, 4, 7, 8, 2, 3, 10, 5, 11, 6, -1, -1, -1,
	9, 2, 1, 9, 10, 2, 9, 4, 10, 7, 10, 4, 5, 11, 6,
	8, 4, 7, 3, 10, 5, 3, 5, 1, 5, 10, 6, -1, -1, -1,
	5, 1, 10, 5, 10, 6, 1, 0, 10, 7, 10, 4, 0, 4, 10,
	0, 5, 9, 0, 6, 5, 0, 3, 6, 10, 6, 3, 8, 4, 7,
	6, 5, 9, 6, 9, 10, 4, 7, 9, 7, 10, 9, -1, -1, -1,
	11, 4, 9, 6, 4, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 11, 6, 4, 9, 11, 0, 8, 3, -1, -1, -1, -1, -1, -1,
	11, 0, 1, 11, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1,
	8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 11, -1, -1, -1,
	1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1,
	3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1,
	0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1,
	11, 4, 9, 11, 6, 4, 10, 2, 3, -1, -1, -1, -1, -1, -1,
	0, 8, 2, 2, 8, 10, 4, 9, 11, 4, 11, 6, -1, -1, -1,
	3, 10, 2, 0, 1, 6, 0, 6, 4, 6, 1, 11, -1, -1, -1,
	6, 4, 1, 6, 1, 11, 4, 8, 1, 2, 1, 10, 8, 10, 1,
	9, 6, 4, 9, 3, 6, 9, 1, 3, 10, 6, 3, -1, -1, -1,
	8, 10, 1, 8, 1, 0, 10, 6, 1, 9, 1, 4, 6, 4, 1,
	3, 10, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1,
	6, 4, 8, 10, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	7, 11, 6, 7, 8, 11, 8, 9, 11, -1, -1, -1, -1, -1, -1,
	0, 7, 3, 0, 11, 7, 0, 9, 11, 6, 7, 11, -1, -1, -1,
	11, 6, 7, 1, 11, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1,
	11, 6, 7, 11, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1,
	1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1,
	2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9,
	7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1,
	7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	2, 3, 10, 11, 6, 8, 11, 8, 9, 8, 6, 7, -1, -1, -1,
	2, 0, 7, 2, 7, 10, 0, 9, 7, 6, 7, 11, 9, 11, 7,
	1, 8, 0, 1, 7, 8, 1, 11, 7, 6, 7, 11, 2, 3, 10,
	10, 2, 1, 10, 1, 7, 11, 6, 1, 6, 7, 1, -1, -1, -1,
	8, 9, 6, 8, 6, 7, 9, 1, 6, 10, 6, 3, 1, 3, 6,
	0, 9, 1, 10, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	7, 8, 0, 7, 0, 6, 3, 10, 0, 10, 6, 0, -1, -1, -1,
	7, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	7, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 0, 8, 10, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 1, 9, 10, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	8, 1, 9, 8, 3, 1, 10, 7, 6, -1, -1, -1, -1, -1, -1,
	11, 1, 2, 6, 10, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 2, 11, 3, 0, 8, 6, 10, 7, -1, -1, -1, -1, -1, -1,
	2, 9, 0, 2, 11, 9, 6, 10, 7, -1, -1, -1, -1, -1, -1,
	6, 10, 7, 2, 11, 3, 11, 8, 3, 11, 9, 8, -1, -1, -1,
	7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1,
	2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1,
	1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1,
	11, 7, 6, 11, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1,
	11, 7, 6, 1, 7, 11, 1, 8, 7, 1, 0, 8, -1, -1, -1,
	0, 3, 7, 0, 7, 11, 0, 11, 9, 6, 11, 7, -1, -1, -1,
	7, 6, 11, 7, 11, 8, 8, 11, 9, -1, -1, -1, -1, -1, -1,
	6, 8, 4, 10, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 6, 10, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1,
	8, 6, 10, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1,
	9, 4, 6, 9, 6, 3, 9, 3, 1, 10, 3, 6, -1, -1, -1,
	6, 8, 4, 6, 10, 8, 2, 11, 1, -1, -1, -1, -1, -1, -1,
	1, 2, 11, 3, 0, 10, 0, 6, 10, 0, 4, 6, -1, -1, -1,
	4, 10, 8, 4, 6, 10, 0, 2, 9, 2, 11, 9, -1, -1, -1,
	11, 9, 3, 11, 3, 2, 9, 4, 3, 10, 3, 6, 4, 6, 3,
	8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1,
	0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1,
	1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1,
	8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 11, 1, -1, -1, -1,
	11, 1, 0, 11, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1,
	4, 6, 3, 4, 3, 8, 6, 11, 3, 0, 3, 9, 11, 9, 3,
	11, 9, 4, 6, 11, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 9, 5, 7, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 8, 3, 4, 9, 5, 10, 7, 6, -1, -1, -1, -1, -1, -1,
	5, 0, 1, 5, 4, 0, 7, 6, 10, -1, -1, -1, -1, -1, -1,
	10, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1,
	9, 5, 4, 11, 1, 2, 7, 6, 10, -1, -1, -1, -1, -1, -1,
	6, 10, 7, 1, 2, 11, 0, 8, 3, 4, 9, 5, -1, -1, -1,
	7, 6, 10, 5, 4, 11, 4, 2, 11, 4, 0, 2, -1, -1, -1,
	3, 4, 8, 3, 5, 4, 3, 2, 5, 11, 5, 2, 10, 7, 6,
	7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1,
	9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1,
	3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1,
	6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8,
	9, 5, 4, 11, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1,
	1, 6, 11, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4,
	4, 0, 11, 4, 11, 5, 0, 3, 11, 6, 11, 7, 3, 7, 11,
	7, 6, 11, 7, 11, 8, 5, 4, 11, 4, 8, 11, -1, -1, -1,
	6, 9, 5, 6, 10, 9, 10, 8, 9, -1, -1, -1, -1, -1, -1,
	3, 6, 10, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1,
	0, 10, 8, 0, 5, 10, 0, 1, 5, 5, 6, 10, -1, -1, -1,
	6, 10, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1,
	1, 2, 11, 9, 5, 10, 9, 10, 8, 10, 5, 6, -1, -1, -1,
	0, 10, 3, 0, 6, 10, 0, 9, 6, 5, 6, 9, 1, 2, 11,
	10, 8, 5, 10, 5, 6, 8, 0, 5, 11, 5, 2, 0, 2, 5,
	6, 10, 3, 6, 3, 5, 2, 11, 3, 11, 5, 3, -1, -1, -1,
	5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1,
	9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1,
	1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8,
	1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 3, 6, 1, 6, 11, 3, 8, 6, 5, 6, 9, 8, 9, 6,
	11, 1, 0, 11, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1,
	0, 3, 8, 5, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	11, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	10, 5, 11, 7, 5, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	10, 5, 11, 10, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1,
	5, 10, 7, 5, 11, 10, 1, 9, 0, -1, -1, -1, -1, -1, -1,
	11, 7, 5, 11, 10, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1,
	10, 1, 2, 10, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1,
	0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 10, -1, -1, -1,
	9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 10, 7, -1, -1, -1,
	7, 5, 2, 7, 2, 10, 5, 9, 2, 3, 2, 8, 9, 8, 2,
	2, 5, 11, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1,
	8, 2, 0, 8, 5, 2, 8, 7, 5, 11, 2, 5, -1, -1, -1,
	9, 0, 1, 5, 11, 3, 5, 3, 7, 3, 11, 2, -1, -1, -1,
	9, 8, 2, 9, 2, 1, 8, 7, 2, 11, 2, 5, 7, 5, 2,
	1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1,
	9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1,
	9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	5, 8, 4, 5, 11, 8, 11, 10, 8, -1, -1, -1, -1, -1, -1,
	5, 0, 4, 5, 10, 0, 5, 11, 10, 10, 3, 0, -1, -1, -1,
	0, 1, 9, 8, 4, 11, 8, 11, 10, 11, 4, 5, -1, -1, -1,
	11, 10, 4, 11, 4, 5, 10, 3, 4, 9, 4, 1, 3, 1, 4,
	2, 5, 1, 2, 8, 5, 2, 10, 8, 4, 5, 8, -1, -1, -1,
	0, 4, 10, 0, 10, 3, 4, 5, 10, 2, 10, 1, 5, 1, 10,
	0, 2, 5, 0, 5, 9, 2, 10, 5, 4, 5, 8, 10, 8, 5,
	9, 4, 5, 2, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	2, 5, 11, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1,
	5, 11, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1,
	3, 11, 2, 3, 5, 11, 3, 8, 5, 4, 5, 8, 0, 1, 9,
	5, 11, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1,
	8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1,
	0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1,
	9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 10, 7, 4, 9, 10, 9, 11, 10, -1, -1, -1, -1, -1, -1,
	0, 8, 3, 4, 9, 7, 9, 10, 7, 9, 11, 10, -1, -1, -1,
	1, 11, 10, 1, 10, 4, 1, 4, 0, 7, 4, 10, -1, -1, -1,
	3, 1, 4, 3, 4, 8, 1, 11, 4, 7, 4, 10, 11, 10, 4,
	4, 10, 7, 9, 10, 4, 9, 2, 10, 9, 1, 2, -1, -1, -1,
	9, 7, 4, 9, 10, 7, 9, 1, 10, 2, 10, 1, 0, 8, 3,
	10, 7, 4, 10, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1,
	10, 7, 4, 10, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1,
	2, 9, 11, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1,
	9, 11, 7, 9, 7, 4, 11, 2, 7, 8, 7, 0, 2, 0, 7,
	3, 7, 11, 3, 11, 2, 7, 4, 11, 1, 11, 0, 4, 0, 11,
	1, 11, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1,
	4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1,
	4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	9, 11, 8, 11, 10, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 0, 9, 3, 9, 10, 10, 9, 11, -1, -1, -1, -1, -1, -1,
	0, 1, 11, 0, 11, 8, 8, 11, 10, -1, -1, -1, -1, -1, -1,
	3, 1, 11, 10, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 2, 10, 1, 10, 9, 9, 10, 8, -1, -1, -1, -1, -1, -1,
	3, 0, 9, 3, 9, 10, 1, 2, 9, 2, 10, 9, -1, -1, -1,
	0, 2, 10, 8, 0, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	3, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	2, 3, 8, 2, 8, 11, 11, 8, 9, -1, -1, -1, -1, -1, -1,
	9, 11, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	2, 3, 8, 2, 8, 11, 0, 1, 8, 1, 11, 8, -1, -1, -1,
	1, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
	-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1
};

__device__ int voxel_indices[][2] =
{
	{ 0, 1 },
	{ 1, 2 },
	{ 2, 3 },
	{ 3, 0 },
	{ 4, 5 },
	{ 5, 6 },
	{ 6, 7 },
	{ 7, 4 },
	{ 0, 4 },
	{ 1, 5 },
	{ 3, 7 },
	{ 2, 6 }
};

struct Voxel
{
	float3 Position;
	float3 Normal;
	float Ambient;
	float Weight;
};

struct VoxelMeshVertex
{
	float3 Position;
	float3 Normal;
	float Ambient;
};

texture<float, 3, hipReadModeElementType> noiseTexture;

extern "C"
{
	inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size)
	{
		uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
		s_Data[pos] = 0;

		pos += size;
		s_Data[pos] = idata;

		for (uint offset = 1; offset < size; offset <<= 1)
		{
			__syncthreads();
			uint t = s_Data[pos] + s_Data[pos - offset];

			__syncthreads();
			s_Data[pos] = t;
		}

		return s_Data[pos];
	}

	inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size)
	{
		return scan1Inclusive(idata, s_Data, size) - idata;
	}

	inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size)
	{
		idata4.y += idata4.x;
		idata4.z += idata4.y;
		idata4.w += idata4.z;

		uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

		idata4.x += oval;
		idata4.y += oval;
		idata4.z += oval;
		idata4.w += oval;

		return idata4;
	}

	inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size)
	{
		uint4 odata4 = scan4Inclusive(idata4, s_Data, size);

		odata4.x -= idata4.x;
		odata4.y -= idata4.y;
		odata4.z -= idata4.z;
		odata4.w -= idata4.w;

		return odata4;
	}

	__global__ void scanExclusiveShared(uint4 *d_Dst, uint4 *d_Src, uint size)
	{
		__shared__ uint s_Data[2 * THREADBLOCK_SIZE];

		uint pos = blockIdx.x * blockDim.x + threadIdx.x;
		uint4 idata4 = d_Src[pos];
		uint4 odata4 = scan4Exclusive(idata4, s_Data, size);

		d_Dst[pos] = odata4;
	}

	__global__ void scanExclusiveShared2(uint *d_Buf, uint *d_Dst, uint *d_Src, uint N, uint arrayLength)
	{
		__shared__ uint s_Data[2 * THREADBLOCK_SIZE];

		uint pos = blockIdx.x * blockDim.x + threadIdx.x;
		uint idata = 0;

		if (pos < N)
		{
			idata = d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] + d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];
		}

		uint odata = scan1Exclusive(idata, s_Data, arrayLength);

		if (pos < N)
		{
			d_Buf[pos] = odata;
		}
	}

	__global__ void uniformUpdate(uint4 *d_Data, uint *d_Buffer)
	{
		__shared__ uint buf;

		uint pos = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadIdx.x == 0)
		{
			buf = d_Buffer[blockIdx.x];
		}

		__syncthreads();

		uint4 data4 = d_Data[pos];

		data4.x += buf;
		data4.y += buf;
		data4.z += buf;
		data4.w += buf;

		d_Data[pos] = data4;
	}

	__global__ void position_weight_noise_cube(Voxel* v, int w, int h, int d)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;
		int cy = h / 2;

		if(x < w && y < h && z < d)
		{
			v[i].Weight = cy - y;
			v[i].Weight += (tex3D(noiseTexture, x / 256.04f, y / 256.01f, z / 255.97f) * 2.0f - 1.0f) * 64.0f;
			v[i].Weight += (tex3D(noiseTexture, x / 128.01f, y / 127.96f, z / 127.98f) * 2.0f - 1.0f) * 4.0f;
			v[i].Weight += (tex3D(noiseTexture, x / 64.01f, y / 64.04f, z / 63.96f) * 2.0f - 1.0f) * 2.0f;
			v[i].Weight += (tex3D(noiseTexture, x / 32.02f, y / 31.98f, z / 31.97f) * 2.0f - 1.0f) * 1.0f;

			v[i].Position.x = x;
			v[i].Position.y = y;
			v[i].Position.z = z;
		}
	}

	__global__ void position_weight_formula(Voxel* v, int w, int h, int d)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;
		
		float area = sqrtf(w * d);
		float3 center = make_float3(w / 2.0f, h / 2.0f, d / 2.0f);
		float3 pillars[3] =
		{
			make_float3(w / 4.0f, 0, d / 4.0f),
			make_float3(w * 3.0f / 4.0f, 0, d * 3.0f / 4.0f),
			make_float3(w * 2.0f / 4.0f, 0, d / 4.0f)
		};

		if(x < w && y < h && z < d)
		{	
			float weight = 0;

			float distanceFromCenter = sqrtf(powf(x - center.x, 2) + powf(z - center.z, 2));
			distanceFromCenter = distanceFromCenter < 0.1f ? 0.1f : distanceFromCenter;

            for(int k = 0; k < 3; k++)
            {
                float distance = sqrtf(powf(x - pillars[k].x, 2) + powf(z - pillars[k].z, 2));
                distance = distance < 0.1f ? 0.1f : distance;
                weight += area / distance;
            }

			weight -= area / distanceFromCenter;

			weight -= powf(distanceFromCenter, 3) / powf(area, 1.5f);

			double coordinate = 3 * HIP_PI * y / h;
			float2 helix = make_float2(cosf(coordinate), sinf(coordinate));
			weight += helix.x * (x - center.x) + helix.y * (z - center.z);

			weight += 10 * cosf(coordinate * 4 / 3);

			weight += (tex3D(noiseTexture, x / 256.04f, y / 256.01f, z / 255.97f) * 2.0f - 1.0f) * 8.0f;
			weight += (tex3D(noiseTexture, x / 128.01f, y / 127.96f, z / 127.98f) * 2.0f - 1.0f) * 4.0f;
			weight += (tex3D(noiseTexture, x / 64.01f, y / 64.04f, z / 63.96f) * 2.0f - 1.0f) * 2.0f;
			weight += (tex3D(noiseTexture, x / 32.02f, y / 31.98f, z / 31.97f) * 2.0f - 1.0f) * 1.0f;

			v[i].Position.x = x;
			v[i].Position.y = y;
			v[i].Position.z = z;
			v[i].Weight = weight;
		}
	}

	__global__ void position_weight_noise_cube_warp(Voxel* v, int w, int h, int d)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;
		int cy = h / 2;

		if(x < w && y < h && z < d)
		{
			float warp = tex3D(noiseTexture, x * 0.004, y * 0.004, z * 0.004);
            float wx = x + warp * 8;
            float wy = y + warp * 8;
            float wz = z + warp * 8;

			v[i].Weight = cy - y;
			v[i].Weight += (tex3D(noiseTexture, wx / 256.04f, wy / 256.01f, wz / 255.97f) * 2.0f - 1.0f) * 64.0f;
			v[i].Weight += (tex3D(noiseTexture, wx / 128.01f, wy / 127.96f, wz / 127.98f) * 2.0f - 1.0f) * 4.0f;
			v[i].Weight += (tex3D(noiseTexture, wx / 64.01f, wy / 64.04f, wz / 63.96f) * 2.0f - 1.0f) * 2.0f;
			v[i].Weight += (tex3D(noiseTexture, wx / 32.02f, wy / 31.98f, wz / 31.97f) * 2.0f - 1.0f) * 1.0f;

			v[i].Position.x = x;
			v[i].Position.y = y;
			v[i].Position.z = z;
		}
	}

	__global__ void normal_ambient(Voxel* v, int w, int h, int d, float ambientRayWidth, int ambientSamplesCount)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;

		if(x < w && y < h && z < d)
		{
			int xii = min(w - 1, x + 1) + y * w + z * w * h;
			int xdi = max(0, x - 1) + y * w + z * w * h;

			int yii = x + min(h - 1, y + 1) * w + z * w * h;
			int ydi = x + max(0, y - 1) * w + z * w * h;

			int zii = x + y * w + min(d - 1, z + 1) * w * h;
			int zdi = x + y * w + max(0, z - 1) * w * h;

			v[i].Normal.x = v[xdi].Weight - v[xii].Weight;
			v[i].Normal.y = v[ydi].Weight - v[yii].Weight;
			v[i].Normal.z = v[zdi].Weight - v[zii].Weight;

			float len = sqrtf(powf(v[i].Normal.x, 2) + powf(v[i].Normal.y, 2) + powf(v[i].Normal.z, 2));

			v[i].Normal.x /= len;
			v[i].Normal.y /= len;
			v[i].Normal.z /= len;

			float stepLength = (w * ambientRayWidth / 100.0f) / ambientSamplesCount;
			float ambient = 0;

			for (int k = 0; k < POISSON_DISC_LEN; k++)
			{
				float sample = 0;

				for (int j = 0; j < ambientSamplesCount; j++)
				{
					int stepNumber = j + 2;

					int cx = (int)fmaxf(0, fminf(w - 1, x + stepNumber * stepLength * poissonDisc[k][0]));
					int cy = (int)fmaxf(0, fminf(h - 1, y + stepNumber * stepLength * poissonDisc[k][1]));
					int cz = (int)fmaxf(0, fminf(d - 1, z + stepNumber * stepLength * poissonDisc[k][2]));

					int ci = cx + cy * w + cz * w * h;

					sample += v[ci].Weight > 0 ? 0 : 1;
				}

				ambient += sample / ambientSamplesCount;
			}

			v[i].Ambient = ambient / POISSON_DISC_LEN;
		}
	}

	__global__ void marching_cubes_cases(Voxel* v, int w, int h, int d, int* offsets, int* triangleCounts, int nW, int nH, int nD)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;
		int wd = w - 1;
		int hd = h - 1;
		int dd = d - 1;
		int id = x + y * wd + z * wd * hd;
		int in = x + y * nW + z * nW * nH;

		if(x < wd && y < hd && z < dd)
		{
			int indices[8];

			indices[0] = i;
			indices[1] = x + y * w + (z + 1) * w * h;
			indices[2] = (x + 1) + y * w + (z + 1) * w * h;
			indices[3] = (x + 1) + y * w + z * w * h;
			indices[4] = x + (y + 1) * w + z * w * h;
			indices[5] = x + (y + 1) * w + (z + 1) * w * h;
			indices[6] = (x + 1) + (y + 1) * w + (z + 1) * w * h;
			indices[7] = (x + 1) + (y + 1) * w + z * w * h;

			int caseNumber = 0;
			for(int k = -1; ++k < 8; caseNumber += v[indices[k]].Weight > 0 ? 1 << k : 0);

			int offset = (255 - caseNumber) * 15;
			offsets[id] = offset;

			int trisCount = 0;
			for (int k = 0; k < 5; k++, offset += 3)
			{
				if (faces[offset] != -1)        
					trisCount++;
				else
					break;
			}

			triangleCounts[in] = trisCount;
		}
	}

	__global__ void marching_cubes_vertices(VoxelMeshVertex* vertices, Voxel* voxels, int* prefixSums, int* offsets, int w, int h, int d, int nW, int nH, int nD)
	{
		int x = threadIdx.x + blockDim.x * blockIdx.x;
		int y = threadIdx.y + blockDim.y * blockIdx.y;
		int z = threadIdx.z + blockDim.z * blockIdx.z;
		int i = x + y * w + z * w * h;
		int wd = w - 1;
		int hd = h - 1;
		int dd = d - 1;
		int id = x + y * wd + z * wd * hd;
		int in = x + y * nW + z * nW * nH;

		if(x < wd && y < hd && z < dd)
		{
			int indices[8];

			indices[0] = i;
			indices[1] = x + y * w + (z + 1) * w * h;
			indices[2] = (x + 1) + y * w + (z + 1) * w * h;
			indices[3] = (x + 1) + y * w + z * w * h;
			indices[4] = x + (y + 1) * w + z * w * h;
			indices[5] = x + (y + 1) * w + (z + 1) * w * h;
			indices[6] = (x + 1) + (y + 1) * w + (z + 1) * w * h;
			indices[7] = (x + 1) + (y + 1) * w + z * w * h;

			bool interpolatedFilled[12] = { false };
			VoxelMeshVertex interpolatedVertices[12];

			for(int k = 0; k < 15; k++)
			{
				int index = faces[offsets[id] + k];

				if(index == -1)
					break;

				if(!interpolatedFilled[index])
				{
					Voxel v1 = voxels[indices[voxel_indices[index][0]]];
					Voxel v2 = voxels[indices[voxel_indices[index][1]]];

					float interpolation = -v1.Weight / (v2.Weight - v1.Weight);

					interpolatedVertices[index].Ambient = v1.Ambient + (v2.Ambient - v1.Ambient) * interpolation;
					interpolatedVertices[index].Position.x = v1.Position.x + (v2.Position.x - v1.Position.x) * interpolation;
					interpolatedVertices[index].Position.y = v1.Position.y + (v2.Position.y - v1.Position.y) * interpolation;
					interpolatedVertices[index].Position.z = v1.Position.z + (v2.Position.z - v1.Position.z) * interpolation;
					interpolatedVertices[index].Normal.x = v1.Normal.x + (v2.Normal.x - v1.Normal.x) * interpolation;
					interpolatedVertices[index].Normal.y = v1.Normal.y + (v2.Normal.y - v1.Normal.y) * interpolation;
					interpolatedVertices[index].Normal.z = v1.Normal.z + (v2.Normal.z - v1.Normal.z) * interpolation;

					interpolatedFilled[index] = true;
				}
			}

			int offset = offsets[id];

			for(int k = 0; k < 5; k++, offset += 3)
			{
				if(faces[offset] == -1)
					break;

				vertices[(prefixSums[in] + k) * 3] = interpolatedVertices[faces[offset]];
				vertices[(prefixSums[in] + k) * 3 + 1] = interpolatedVertices[faces[offset + 1]];
				vertices[(prefixSums[in] + k) * 3 + 2] = interpolatedVertices[faces[offset + 2]];
			}
		}
	}
}